#include "hip/hip_runtime.h"
#include "adaptor.h"
#include "comm.h"
#include "flagcx.h"
#include "flagcx_kernel.h"

__device__ __forceinline__ void spin_backoff(int iter) {
  int delay = 1 << (iter < 15 ? iter : 15);
#if __CUDA_ARCH__ >= 700
  __nanosleep(delay);
#else
  uint64_t start = clock64();
  while (clock64() - start < (uint64_t)delay) { /* spin */
  }
#endif
}

__device__ size_t getFlagcxDataTypeSizeDevice(flagcxDataType_t dtype) {
  switch (dtype) {
    // case flagcxInt8:
    case flagcxChar:
      return sizeof(char); // 1 byte
    case flagcxUint8:
      return sizeof(unsigned char); // 1 byte
    // case flagcxInt32:
    case flagcxInt:
      return sizeof(int); // 4 bytes
    case flagcxUint32:
      return sizeof(unsigned int); // 4 bytes
    case flagcxInt64:
      return sizeof(long long); // 8 bytes
    case flagcxUint64:
      return sizeof(unsigned long long); // 8 bytes
    // case flagcxFloat16:
    case flagcxHalf:
      return 2; // Half precision float is 2 bytes
    // case flagcxFloat32:
    case flagcxFloat:
      return sizeof(float); // 4 bytes
    // case flagcxFloat64:
    case flagcxDouble:
      return sizeof(double); // 8 bytes
    case flagcxBfloat16:
      return 2; // BFloat16 is typically 2 bytes
    default:
      return 0;
  }
}

__device__ flagcxResult_t flagcxDeviceSend(const void *sendbuff, size_t count,
                                           flagcxDataType_t datatype, int peer,
                                           flagcxFifo_t fifo) {
  flagcxDeviceTrigger trigger;
  trigger.fields.addr = (uint64_t)(sendbuff);
  trigger.fields.count = count;
  trigger.fields.peerRank = peer;
  trigger.fields.datatype = datatype;
  trigger.fields.type = flagcxDevicePrimSend;
  trigger.fields.ready = 0;
  fifo->enqueue(trigger);
  return flagcxSuccess;
}

__device__ flagcxResult_t flagcxDeviceRecv(void *recvbuff, size_t count,
                                           flagcxDataType_t datatype, int peer,
                                           flagcxFifo_t fifo) {
  flagcxDeviceTrigger trigger;
  trigger.fields.addr = (uint64_t)(recvbuff);
  trigger.fields.count = count;
  trigger.fields.peerRank = peer;
  trigger.fields.datatype = datatype;
  trigger.fields.type = flagcxDevicePrimRecv;
  trigger.fields.ready = 0;
  fifo->enqueue(trigger);
  return flagcxSuccess;
}

__device__ flagcxResult_t flagcxDeviceWait(flagcxFifo_t fifo) {
  int curr_p = __ldg(fifo->produced);
  int curr_c = __ldg(fifo->consumed);
  int iter = 0;
  while (curr_p > curr_c) {
    // curr_p = __ldg(comm->proxyKernelState->fifo->produced);
    // check a fixed point, not updating `produced` index
    curr_c = __ldg(fifo->consumed);
    spin_backoff(iter);
    iter++;
  }
  return flagcxSuccess;
}

__device__ flagcxResult_t flagcxDeviceTerm(flagcxFifo_t fifo) {
  flagcxDeviceTrigger trigger;
  trigger.fields.addr = 0;
  trigger.fields.count = 0;
  trigger.fields.peerRank = 0;
  trigger.fields.datatype = 0;
  trigger.fields.type = flagcxDevicePrimTerm;
  trigger.fields.ready = 0;
  fifo->enqueue(trigger);
  return flagcxSuccess;
}

__host__ flagcxResult_t flagcxFifo::initFifo(int32_t capacity_) {
  TRACE(FLAGCX_P2P, "Initialize FIFO...");
  void *tmp_buff;
  deviceAdaptor->deviceMalloc(&tmp_buff, capacity * sizeof(flagcxDeviceTrigger),
                              flagcxMemHost, NULL);
  buffer = (uint64_t *)tmp_buff;
  void *tmp_p;
  deviceAdaptor->deviceMalloc(&tmp_p, sizeof(int32_t), flagcxMemHost, NULL);
  produced = (int32_t *)tmp_p;
  void *tmp_c;
  deviceAdaptor->deviceMalloc(&tmp_c, sizeof(int32_t), flagcxMemHost, NULL);
  consumed = (int32_t *)tmp_c;
  void *tmp_term;
  deviceAdaptor->deviceMalloc(&tmp_term, sizeof(int32_t), flagcxMemHost, NULL);
  terminate = (int32_t *)tmp_term;
  produced[0] = -1;
  consumed[0] = -1;
  terminate[0] = -1;
  return flagcxSuccess;
}

__host__ flagcxResult_t flagcxFifo::freeFifo() {
  deviceAdaptor->deviceFree(buffer, flagcxMemHost, NULL);
  deviceAdaptor->deviceFree(produced, flagcxMemHost, NULL);
  deviceAdaptor->deviceFree(consumed, flagcxMemHost, NULL);
  deviceAdaptor->deviceFree(terminate, flagcxMemHost, NULL);
  return flagcxSuccess;
}

__device__ flagcxResult_t flagcxFifo::enqueue(flagcxDeviceTrigger trigger) {
  int idx = -1;
  while (true) {
    int old_c = consumed[0];
    int old_p = produced[0];
    if (old_p - old_c < capacity) {
      int prev = atomicCAS(produced, old_p, old_p + 1);
      if (prev == old_p) {
        idx = (old_p + 1) % capacity;
        break;
      }
    }
  }
  *(buffer + 2 * idx) = trigger.value.fst;
  *(buffer + 2 * idx + 1) = trigger.value.snd;
  *(buffer + 2 * idx + 1) |= 1ULL;
  return flagcxSuccess;
}

__host__ flagcxResult_t flagcxFifo::dequeue(flagcxDeviceTrigger_t trigger) {
  /*
  int idx = -1;
  while (true) {
    int old_c = consumed[0];
    int old_p = produced[0];
    if (old_c < old_p) {
      int prev = atomicCAS(consumed, old_c, old_c + 1);
      if (prev == old_c) {
        idx = old_c + 1;
        break;
      }
    }
  }
  */
  int idx = -1;
  int old_c = consumed[0];
  int old_p = produced[0];
  while (true) {
    if (old_p < produced[0]) {
      TRACE(FLAGCX_P2P, "fifo dequeue loop, produced = %d", old_p);
    }
    if (old_p > old_c) {
      idx = (old_c + 1) % capacity;
      while (true) {
        // check if producer write is complete
        TRACE(FLAGCX_P2P, "write check loop");
        if (*(buffer + 2 * idx + 1) & 1ULL) {
          break;
        }
      }
      break;
    }
    old_c = consumed[0];
    old_p = produced[0];
   }
  (*trigger).value.fst = *(buffer + 2 * idx);
  (*trigger).value.snd = *(buffer + 2 * idx + 1);
  *(buffer + 2 * idx) = 0ULL;
  *(buffer + 2 * idx + 1) = 0ULL;
  consumed[0]++;
  return flagcxSuccess;
}

__host__ flagcxResult_t flagcxFifo::enqueue(flagcxReduceTrigger trigger) {
  // to be implemented
  return flagcxNotSupported;
}

__device__ flagcxResult_t flagcxFifo::dequeue(flagcxReduceTrigger_t trigger) {
  // to be implemented
  return flagcxNotSupported;
}
