#include "hip/hip_runtime.h"
#include "flagcx_kernel.h"
#include "global_comm.h"

__global__ void flagcxP2pDemo(const void *sendbuff, void *recvbuff,
                              size_t count, flagcxDataType_t datatype,
                              int sendPeer, int recvPeer, flagcxComm_t comm) {
  int tid = threadIdx.x;
  if (tid == 0) {
    for (int i = 0; i < 16; i++) {
      const void *sendaddr = static_cast<const void *>(
          static_cast<char *>(const_cast<void *>(sendbuff)) +
          count / 16 * i * getFlagcxDataTypeSizeDevice(datatype));
      flagcxDeviceSend(sendaddr, count / 16, datatype, sendPeer,
                       comm->hetero_comm);
    }
    for (int i = 0; i < 16; i++) {
      void *recvaddr = static_cast<void *>(
          static_cast<char *>(recvbuff) +
          count / 16 * i * getFlagcxDataTypeSizeDevice(datatype));
      flagcxDeviceRecv(recvaddr, count / 16, datatype, recvPeer,
                       comm->hetero_comm);
    }
    flagcxDeviceTerm(comm->hetero_comm);
    flagcxDeviceWait(comm->hetero_comm);
  }
}
