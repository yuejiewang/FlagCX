#include "hip/hip_runtime.h"
#include "flagcx.h"
#include "flagcx_kernel.h"
#include "global_comm.h"
#include "nvidia_adaptor.h"
__global__ void flagcxP2pKernel(const void *sendbuff, void *recvbuff,
                              size_t count, flagcxDataType_t datatype,
                              int sendPeer, int recvPeer, flagcxComm_t comm) {
  int tid = threadIdx.x;
  if (tid == 0) {
    for (int i = 0; i < 16; i++) {
      const void *sendaddr = static_cast<const void *>(
          static_cast<char *>(const_cast<void *>(sendbuff)) +
          count / 16 * i * getFlagcxDataTypeSizeDevice(datatype));
      flagcxDeviceSend(sendaddr, count / 16, datatype, sendPeer,
                       comm->hetero_comm);
    }
    for (int i = 0; i < 16; i++) {
      void *recvaddr = static_cast<void *>(
          static_cast<char *>(recvbuff) +
          count / 16 * i * getFlagcxDataTypeSizeDevice(datatype));
      flagcxDeviceRecv(recvaddr, count / 16, datatype, recvPeer,
                       comm->hetero_comm);
    }
    flagcxDeviceTerm(comm->hetero_comm);
    flagcxDeviceWait(comm->hetero_comm);
  }
}

void flagcxP2pDemo(const void *sendbuff, void *recvbuff, size_t count,
                            flagcxDataType_t datatype, int sendPeer,
                            int recvPeer, flagcxComm_t comm,
                            flagcxStream_t stream) {
  flagcxP2pKernel<<<1, 1, 0, stream->base>>>(
      sendbuff, recvbuff, count, datatype, sendPeer, recvPeer, comm);
}
